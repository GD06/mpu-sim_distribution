#include "hip/hip_runtime.h"
#ifndef _BLUR_CUDA_KERNEL
#define _BLUR_CUDA_KERNEL 

#include "cta_config.h"

__global__ void Blur3x3(float* input, float* output, 
        int num_rows, int num_cols, float alpha) {
    int bidx = blockIdx.x;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    int num_matrix_blocks = (num_rows * num_cols) / (BLOCK_SIZE * BLOCK_SIZE);

    __shared__ float tmp_buffer[BLOCK_SIZE][BLOCK_SIZE];

    for (int block_id = bidx; block_id < num_matrix_blocks; block_id += gridDim.x) {
        for (int y = 0; y < BLOCK_SIZE; y += NUM_THREADS_Y) {
            tmp_buffer[y + tidy][tidx] = input[
                (y / NUM_THREADS_Y) * NUM_THREADS_Y * NUM_THREADS_X * num_matrix_blocks
                + block_id * NUM_THREADS_Y * NUM_THREADS_X
                + tidy * NUM_THREADS_X + tidx];
        }
        __syncthreads();

        for (int y = 0; y < BLOCK_SIZE; y += NUM_THREADS_Y) {
            float sum_val = 0.0f;
            for (int ky = 0; ky < 3; ++ky) {
                for (int kx = 0; kx < 3; ++kx) {
                    int row_index = y + tidy + ky - 1;
                    int col_index = tidx + kx - 1;
                    CLAMP(row_index, 0, BLOCK_SIZE);
                    CLAMP(col_index, 0, BLOCK_SIZE);
                    sum_val += tmp_buffer[row_index][col_index];
                }
            }

            output[
                (y / NUM_THREADS_Y) * NUM_THREADS_Y * NUM_THREADS_X * num_matrix_blocks
                + block_id * NUM_THREADS_Y * NUM_THREADS_X 
                + tidy * NUM_THREADS_X + tidx] = sum_val * alpha;
        }
        __syncthreads(); 
    }

    return;
}

#endif 
