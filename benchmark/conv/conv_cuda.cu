#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include "cta_config.h"
#include "../common/cuda_check.h" 

extern __global__ void Conv3x3(
        float* input, float* kernel, float* output, int num_rows, int num_cols);

void RandFloatArray(float* ptr, int length) {
    for (int i = 0; i < length; ++i) {
        float val = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        ptr[i] = val; 
    }
    return; 
}

void AssertArrayEqual(float* ptr1, float* ptr2, int length, float precision = 1e-5) {
    for (int i = 0; i < length; ++i) {
        assert(fabs(ptr1[i] - ptr2[i]) < precision);
    }
    return;
}

int main(int argc, char** argv) {
    if (argc < 3) {
        printf("Usage: ./conv <num of rows> <num of columns>");
        return -1;
    }

    int num_rows = atoi(argv[1]);
    int num_cols = atoi(argv[2]);
    printf("Running the 3x3 conv for an input size %d x %d\n",
            num_rows, num_cols);

    float* host_input = (float*) malloc(num_rows * num_cols * sizeof(float));
    float* host_kernel = (float*) malloc(9 * sizeof(float));
    float* host_output = (float*) malloc(num_rows * num_cols * sizeof(float));

    RandFloatArray(host_input, num_rows * num_cols);
    RandFloatArray(host_kernel, 9); 

    int num_matrix_blocks = (num_rows * num_cols) / (BLOCK_SIZE * BLOCK_SIZE);
    for (int bid = 0; bid < num_matrix_blocks; ++bid) {
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                float sum_val = 0.0f;
                for (int ki = 0; ki < 3; ++ki) {
                    for (int kj = 0; kj < 3; ++kj) {
                        int row_index = i + ki - 1;
                        int col_index = j + kj - 1;
                        CLAMP(row_index, 0, BLOCK_SIZE);
                        CLAMP(col_index, 0, BLOCK_SIZE);

                        int slice_id = (row_index / NUM_THREADS_Y);
                        int row_id = (row_index % NUM_THREADS_Y);
                        sum_val += (host_input[
                            slice_id * NUM_THREADS_X * NUM_THREADS_Y * num_matrix_blocks
                            + bid * NUM_THREADS_X * NUM_THREADS_Y
                            + row_id * NUM_THREADS_X + col_index] * host_kernel[
                            ki * 3 + kj]);
                    }
                }

                int dst_slice_id = (i / NUM_THREADS_Y);
                int dst_row_id = (i % NUM_THREADS_Y);
                host_output[
                    dst_slice_id * NUM_THREADS_X * NUM_THREADS_Y * num_matrix_blocks
                    + bid * NUM_THREADS_X * NUM_THREADS_Y
                    + dst_row_id * NUM_THREADS_X + j] = sum_val;
            }
        }
    }
    printf("Completed ground truth computation!\n");

    float* device_input;
    float* device_kernel;
    float* device_output;

    CUDA_CHECK(hipMalloc((void**) &device_input, 
                num_rows * num_cols * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**) &device_kernel, 9 * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**) &device_output,
                num_rows * num_cols * sizeof(float)));
    float* results = (float*) malloc(num_rows * num_cols * sizeof(float));

    CUDA_CHECK(hipMemcpy(device_input, host_input,
                num_rows * num_cols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(device_kernel, host_kernel,
                9 * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(NUM_THREADS_X, NUM_THREADS_Y);

#ifdef MEASURE_POWER 
    while (true) {
#endif 

    Conv3x3<<<NUM_BLOCKS, dimBlock>>>(device_input, device_kernel, device_output, 
            num_rows, num_cols);
    hipDeviceSynchronize();

#ifdef MEASURE_POWER 
    }
#endif 

    printf("Completed GPU computation!\n");

    CUDA_CHECK(hipMemcpy(results, device_output, 
                num_rows * num_cols * sizeof(float), hipMemcpyDeviceToHost));

    AssertArrayEqual(host_output, results, num_rows * num_cols);
    printf("Correctness Check: Accepted!\n");

    free(host_input);
    free(host_kernel);
    free(host_output);
    free(results);

    CUDA_CHECK(hipFree(device_input));
    CUDA_CHECK(hipFree(device_kernel));
    CUDA_CHECK(hipFree(device_output));
    
    return 0; 
}
